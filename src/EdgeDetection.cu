#include <algorithm>
#include <iostream>
#include <stdio.h>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"
#include "ImageProc.h"
#include "ImageUtils.h"

/**
 * Parses the command-line arguments
 *
 * @param argc The number of command-line arguments
 * @param argv The command-line arguments
 *
 * @return Flag denoting success or failure
 */
__host__
int32_t parseCmdArgs(int32_t argc, char ** argv, std::string & fileName)
{
    if(argc != 2)
    {
        printf("Usage:\n");
        printf("edge_detection.exe fileName\n");

        return EXIT_FAILURE;
    }
    else
    {
        fileName = std::string(argv[1]);
    }

    return EXIT_SUCCESS;
}

/**
 * Converts pixel data to grayscale and applies a filter using the CPU or GPU
 *
 * @param format      The file format
 * @param imageWidth  The width of the image to write
 * @param imageHeight The height of the image to write
 * @param filter      The type of image filter to use
 * @param pixelData   The channel-separated pixel data
 * @param useCPU      Flag denoting whether to use the CPU or GPU
 *
 * @return Flag denoting success or failure
 */
__host__
int32_t applyFilterGray(const FREE_IMAGE_FORMAT & format,
                        uint32_t imageWidth,
                        uint32_t imageHeight,
                        uint32_t bitsPerPixel,
                        ImageFilter filter, 
                        const BYTE * pixelData, 
                        bool useCPU)
{
    uint32_t imageSize = imageWidth * imageHeight;

    // Allocate grayscale pixel memory
    BYTE * grayPixelData;
    
    if(useCPU)
    {
        grayPixelData = new BYTE[imageSize];
    }
    else
    {
        checkCudaErrors(hipHostMalloc((void **) &grayPixelData, sizeof(BYTE) * imageSize, hipHostMallocDefault));
    }

    // Convert RGB pixel data to grayscale
    int32_t status = rgbToGray(imageWidth, imageHeight, bitsPerPixel, pixelData, grayPixelData);

    if(status == EXIT_FAILURE)
    {
        return EXIT_FAILURE;
    }

    // Apply filter
    status = applyFilter(format, imageWidth, imageHeight, filter, grayPixelData, useCPU);

    // Cleanup
    if(useCPU)
    {
        delete [] grayPixelData;
    }
    else
    {
        checkCudaErrors(hipHostFree(grayPixelData));
    }

    return EXIT_SUCCESS;
}

/**
 * Entry point to the application
 *
 * @param argc The number of command-line arguments
 * @param argv The command-line arguments
 *
 * @return Exit code indicating success or failure
 */
int32_t main(int32_t argc, char ** argv)
{
    /*
     * Parse command-line arguments
     */
    std::string fileName;

    int32_t status = parseCmdArgs(argc, argv, fileName);

    if(status == EXIT_FAILURE)
    {
        return EXIT_FAILURE;
    }

    /*
     * Load the image data
     */
    FREE_IMAGE_FORMAT format;
    FIBITMAP * bitmap = NULL;

    status = loadImage(fileName, format, &bitmap);

    if(status == EXIT_FAILURE)
    {
        return EXIT_FAILURE;
    }

    /*
     * Get the image info
     */
    uint32_t imageWidth;
    uint32_t imageHeight;
    uint32_t bitsPerPixel;

    status = getImageInfo(&bitmap, imageWidth, imageHeight, bitsPerPixel);

    if(status == EXIT_FAILURE)
    {
        FreeImage_Unload(bitmap);
        return EXIT_FAILURE;
    }

    /*
     * Load the pixel data
     */
    BYTE * pixelData;
    
    // Check if 8 or 24 bits per pixel
    if(bitsPerPixel == 8 || bitsPerPixel == 24)
    {
        pixelData = new BYTE[imageWidth * imageHeight * (bitsPerPixel / 8)];
    }
    // Check if 32 bits per pixel
    else if(bitsPerPixel == 32)
    {
        pixelData = new BYTE[imageWidth * imageHeight * 3];
    }
    // Unsupported pixel format
    else
    {
        std::cerr << "Unsupported pixel format!" << std::endl;
        
        return EXIT_FAILURE;
    }

    status = loadPixelData(&bitmap, imageWidth, imageHeight, bitsPerPixel, pixelData);

    if(status == EXIT_FAILURE)
    {
        return EXIT_FAILURE;
    }

    /*
     * Apply Sobel filter to RGB pixel values
     */
    // Apply Sobel filter using CPU
    status = applyFilterGray(format, imageWidth, imageHeight, bitsPerPixel, SOBEL, pixelData, true);

    if(status == EXIT_FAILURE)
    {
        return EXIT_FAILURE;
    }

    // Apply Sobel filter using GPU

    /*
     * Apply Sobel filter to HSV channels
     */
    // Apply Sobel filter using CPU

    // Apply Sobel filter using GPU

    /*
     * Cleanup
     */
    FreeImage_Unload(bitmap);

    delete [] pixelData;

    return EXIT_SUCCESS;
}