#include <algorithm>
#include <iostream>
#include <stdio.h>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"
#include "ImageUtils.h"
#include "Sobel.h"

/**
 * Parses the command-line arguments
 *
 * @param argc The number of command-line arguments
 * @param argv The command-line arguments
 *
 * @return Flag denoting success or failure
 */
__host__
int32_t parseCmdArgs(int32_t argc, char ** argv, std::string & fileName)
{
    if(argc != 2)
    {
        printf("Usage:\n");
        printf("edge_detection.exe fileName\n");

        return EXIT_FAILURE;
    }
    else
    {
        fileName = std::string(argv[1]);
    }

    return EXIT_SUCCESS;
}

/**
 * Apply a Sobel filter to grayscale values
 *
 * @param format       The file format
 * @param imageWidth   The width of the image to write
 * @param imageHeight  The height of the image to write
 * @param bitsPerPixel The bits per pixel of the image to write
 * @param pixelData    The channel separated RGB pixel data
 * @param useCPU       Flag denoting whether to use CPU or GPU
 *
 * @return Flag denoting success or failure
 */
int32_t applySobelFilterGrayscale(const FREE_IMAGE_FORMAT & format, 
                                  uint32_t imageWidth, 
                                  uint32_t imageHeight, 
                                  uint32_t bitsPerPixel, 
                                  const BYTE * pixelData, 
                                  bool useCPU)
{
    // Convert pixel data to grayscale
    uint32_t imageSize = imageWidth * imageHeight;

    BYTE * h_grayPixelData = new BYTE[imageSize];
    BYTE * h_outputPixelData = new BYTE[imageSize];

    if(bitsPerPixel == 8)
    {
        memcpy(h_grayPixelData, pixelData, imageWidth * imageHeight * sizeof(BYTE));
    }
    else
    {
        for(uint32_t y = 0; y < imageHeight; y++)
        {
            for(uint32_t x = 0; x < imageWidth; x++)
            {
                BYTE r = pixelData[(y * imageWidth) + x];
                BYTE g = pixelData[(y * imageWidth) + imageSize + x];
                BYTE b = pixelData[(y * imageWidth) + (imageSize * 2) + x];

                h_grayPixelData[(y * imageWidth) + x] = (r + g + b) / 3;
            }
        }
    }

    // Apply Sobel filter using CPU
    if(useCPU)
    {
        applySobelFilterCPU(imageWidth, imageHeight, h_grayPixelData, h_outputPixelData);
    }
    // Apply Sobel filter using GPU
    else
    {
        //TODO: Setup device memory
        //TODO: Execute kernel
        //TODO: Cleanup device memory
    }

    // Output results
    std::string outputFileName = "grayscale_output.png";

    int32_t status = saveImage(outputFileName, format, imageWidth, imageHeight, 8, h_outputPixelData);

    if(status == EXIT_FAILURE)
    {
        return EXIT_FAILURE;
    }

    // Cleanup
    delete [] h_grayPixelData;
    delete [] h_outputPixelData;

    return EXIT_SUCCESS;
}

/**
 * Entry point to the application
 *
 * @param argc The number of command-line arguments
 * @param argv The command-line arguments
 *
 * @return Exit code indicating success or failure
 */
int32_t main(int32_t argc, char ** argv)
{
    /*
     * Parse command-line arguments
     */
    std::string fileName;

    int32_t status = parseCmdArgs(argc, argv, fileName);

    if(status == EXIT_FAILURE)
    {
        return EXIT_FAILURE;
    }

    /*
     * Load the image data
     */
    FREE_IMAGE_FORMAT format;
    FIBITMAP * bitmap = NULL;

    status = loadImage(fileName, format, &bitmap);

    if(status == EXIT_FAILURE)
    {
        return EXIT_FAILURE;
    }

    /*
     * Get the image info
     */
    uint32_t imageWidth;
    uint32_t imageHeight;
    uint32_t bitsPerPixel;

    status = getImageInfo(&bitmap, imageWidth, imageHeight, bitsPerPixel);

    if(status == EXIT_FAILURE)
    {
        FreeImage_Unload(bitmap);
        return EXIT_FAILURE;
    }

    /*
     * Load the pixel data
     */
    BYTE * pixelData;
    
    // Check if 8 or 24 bits per pixel
    if(bitsPerPixel == 8 || bitsPerPixel == 24)
    {
        pixelData = new BYTE[imageWidth * imageHeight * (bitsPerPixel / 8)];
    }
    // Check if 32 bits per pixel
    else if(bitsPerPixel == 32)
    {
        pixelData = new BYTE[imageWidth * imageHeight * 3];
    }
    // Unsupported pixel format
    else
    {
        std::cerr << "Unsupported pixel format!" << std::endl;
        
        return EXIT_FAILURE;
    }

    status = loadPixelData(&bitmap, imageWidth, imageHeight, bitsPerPixel, pixelData);

    if(status == EXIT_FAILURE)
    {
        return EXIT_FAILURE;
    }

    /*
     * Apply Sobel filter to grayscale values
     */
    // Apply Sobel filter using CPU
    status = applySobelFilterGrayscale(format, imageWidth, imageHeight, bitsPerPixel, pixelData, true);

    if(status == EXIT_FAILURE)
    {
        return EXIT_FAILURE;
    }

    // Apply Sobel filter using GPU

    /*
     * Apply Sobel filter to HSV channels
     */
    // Apply Sobel filter using CPU

    // Apply Sobel filter using GPU

    /*
     * Cleanup
     */
    FreeImage_Unload(bitmap);

    delete [] pixelData;

    return EXIT_SUCCESS;
}