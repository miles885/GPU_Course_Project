#include "hip/hip_runtime.h"
#include "ImageProc.h"

#include "hip/hip_runtime_api.h"

//NOTE: The sobel algorithm and kernels were found on
//      https://en.wikipedia.org/wiki/Sobel_operator
//      under the "Pseduocode implementation" section

/**
 * Creates an image with highlighted edges by
 * applying a filter to the pixel data
 *
 * @param imageWidth      The width of the image to write
 * @param imageHeight     The height of the image to write
 * @param filterX         The x dimension filter data
 * @param filterY         The y dimension filter data
 * @param pixelData       The input array of pixel data
 * @param outputPixelData The output array of pixel data
 *
 * @return None
 */
 __host__
void applyFilterCPU(uint32_t imageWidth, 
                    uint32_t imageHeight, 
                    const int32_t * filterX, 
                    const int32_t * filterY, 
                    const BYTE * pixelData, 
                    BYTE * outputPixelData)
{
    for(uint32_t y = 1; y <= (imageHeight - 2); y++)
    {
        for(uint32_t x = 1; x <= (imageWidth - 2); x++)
        {
            int32_t topRow[3] = {pixelData[((y - 1) * imageWidth) + (x - 1)], pixelData[((y - 1) * imageWidth) + x], pixelData[((y - 1) * imageWidth) + (x + 1)]};
            int32_t midRow[3] = {pixelData[(y * imageWidth) + (x - 1)],       pixelData[(y * imageWidth) + x],       pixelData[(y * imageWidth) + (x + 1)]};
            int32_t botRow[3] = {pixelData[((y + 1) * imageWidth) + (x - 1)], pixelData[((y + 1) * imageWidth) + x], pixelData[((y + 1) * imageWidth) + (x + 1)]};

            int32_t pixelX = (filterX[0] * topRow[0]) + (filterX[1] * topRow[1]) + (filterX[2] * topRow[2]) +
                             (filterX[3] * midRow[0]) + (filterX[4] * midRow[1]) + (filterX[5] * midRow[2]) +
                             (filterX[6] * botRow[0]) + (filterX[7] * botRow[1]) + (filterX[8] * botRow[2]);
            
            int32_t pixelY = (filterY[0] * topRow[0]) + (filterY[1] * topRow[1]) + (filterY[2] * topRow[2]) +
                             (filterY[3] * midRow[0]) + (filterY[4] * midRow[1]) + (filterY[5] * midRow[2]) +
                             (filterY[6] * botRow[0]) + (filterY[7] * botRow[1]) + (filterY[8] * botRow[2]);
            
            // Calculate magnitude
            int32_t mag = sqrt((pixelX * pixelX) + (pixelY * pixelY));

            // Set output pixel value
            //TODO: Use some pixel threshold for better results?
            outputPixelData[(y * imageWidth) + x] = mag;
        }
    }
}

/**
 * Creates an image with highlighted edges by
 * applying a filter to the pixel data
 *
 * @param imageWidth      The width of the image to write
 * @param imageHeight     The height of the image to write
 * @param filterX         The x dimension filter data
 * @param filterY         The y dimension filter data
 * @param pixelData       The input array of pixel data
 * @param outputPixelData The output array of pixel data
 *
 * @return None
 */
 __global__
void applyFilterGPU(uint32_t imageWidth, 
                    uint32_t imageHeight, 
                    const int32_t * filterX, 
                    const int32_t * filterY, 
                    const BYTE * pixelData, 
                    BYTE * outputPixelData)
{

}

/**
 * Apply a filter to a set of pixel values using the CPU or GPU
 *
 * @param format      The file format
 * @param imageWidth  The width of the image to write
 * @param imageHeight The height of the image to write
 * @param filter      The type of image filter to use
 * @param pixelData   The single channel pixel data
 * @param useCPU      Flag denoting whether to use the CPU or GPU
 *
 * @return Flag denoting success or failure
 */
__host__
int32_t applyFilter(const FREE_IMAGE_FORMAT & format, 
                    uint32_t imageWidth, 
                    uint32_t imageHeight, 
                    ImageFilter filter, 
                    const BYTE * pixelData, 
                    bool useCPU)
{
    uint32_t imageSize = imageWidth * imageHeight;

    // Host memory
    BYTE * h_outputPixelData;
    int32_t * h_filterX;
    int32_t * h_filterY;

    // Set filter data
    switch (filter)
    {
        case SOBEL:
            h_filterX = const_cast<int32_t *>(SOBEL_X);
            h_filterY = const_cast<int32_t *>(SOBEL_Y);
            break;
        default:
            break;
    }

    // Check to see if using the CPU
    if(useCPU)
    {
        // Allocate memory
        h_outputPixelData = new BYTE[imageSize];

        // Apply filter
        applyFilterCPU(imageWidth, imageHeight, h_filterX, h_filterY, pixelData, h_outputPixelData);
    }
    // Using the GPU
    else
    {
        uint32_t imageSizeBytes = sizeof(BYTE) * imageSize;

        // Allocate host memory
        checkCudaErrors(hipHostMalloc((void **) &h_outputPixelData, imageSizeBytes, hipHostMallocDefault));

        // Allocate device memory
        BYTE * d_pixelData;
        BYTE * d_outputPixelData;

        checkCudaErrors(hipMalloc((void **) &d_pixelData, imageSizeBytes));
        checkCudaErrors(hipMalloc((void **) &d_outputPixelData, imageSizeBytes));

        // Copy pixel data to device
        hipMemcpy(d_pixelData, pixelData, imageSizeBytes, hipMemcpyHostToDevice);

        //TODO: Execute kernel

        // Cleanup
        checkCudaErrors(hipFree(d_pixelData));
        checkCudaErrors(hipFree(d_outputPixelData));
    }

    // Output results
    std::string outputFileName = "sobel_output.png";

    int32_t status = saveImage(outputFileName, format, imageWidth, imageHeight, 8, h_outputPixelData);

    if(status == EXIT_FAILURE)
    {
        return EXIT_FAILURE;
    }

    // Cleanup
    if(useCPU)
    {
        delete[] h_outputPixelData;
    }
    else
    {
        hipHostFree(h_outputPixelData);
    }

    return EXIT_SUCCESS;
 }